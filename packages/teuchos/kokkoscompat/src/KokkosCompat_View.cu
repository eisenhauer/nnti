
#include <hip/hip_runtime.h>
/*
//@HEADER
// ************************************************************************
//
//                             Kokkos
//         Manycore Performance-Portable Multidimensional Arrays
//
//              Copyright (2012) Sandia Corporation
//
// Under the terms of Contract DE-AC04-94AL85000 with Sandia Corporation,
// the U.S. Government retains certain rights in this software.
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions are
// met:
//
// 1. Redistributions of source code must retain the above copyright
// notice, this list of conditions and the following disclaimer.
//
// 2. Redistributions in binary form must reproduce the above copyright
// notice, this list of conditions and the following disclaimer in the
// documentation and/or other materials provided with the distribution.
//
// 3. Neither the name of the Corporation nor the names of the
// contributors may be used to endorse or promote products derived from
// this software without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY SANDIA CORPORATION "AS IS" AND ANY
// EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
// PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL SANDIA CORPORATION OR THE
// CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
// EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
// PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
// PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF
// LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING
// NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
// SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
//
// Questions?  Contact  H. Carter Edwards (hcedwar@sandia.gov)
//
// ************************************************************************
//@HEADER
*/

#if 0

#include "KokkosCompat_View.hpp"
#include "KokkosCompat_View_def.hpp"

#include "Kokkos_Core.hpp"

namespace Kokkos {
  namespace Compat {

#define COMPAT_INSTANT_CUDA(T) \
    COMPAT_INSTANT(T,Kokkos::Cuda)

#define COMPAT_INSTANT_CUDA_UVM(T) \
    COMPAT_INSTANT(T,Kokkos::CudaUVMSpace)

    COMPAT_INSTANT_CUDA(float)
    COMPAT_INSTANT_CUDA(double)
    COMPAT_INSTANT_CUDA(int)
    COMPAT_INSTANT_CUDA(long)
    COMPAT_INSTANT_CUDA(unsigned)
    COMPAT_INSTANT_CUDA(unsigned long)
    COMPAT_INSTANT_CUDA(char)
    COMPAT_INSTANT_CUDA(short)

    COMPAT_INSTANT_CUDA_UVM(float)
    COMPAT_INSTANT_CUDA_UVM(double)
    COMPAT_INSTANT_CUDA_UVM(int)
    COMPAT_INSTANT_CUDA_UVM(long)
    COMPAT_INSTANT_CUDA_UVM(unsigned)
    COMPAT_INSTANT_CUDA_UVM(unsigned long)
    COMPAT_INSTANT_CUDA_UVM(char)
    COMPAT_INSTANT_CUDA_UVM(short)

  } // namespace Compat
} // namespace Kokkos

#endif // 0
